#include <pybind11/pybind11.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>

hipdnnHandle_t cudnn;

void init_cudnn() {
    hipdnnCreate(&cudnn);
}

float cudnn_add(float a, float b) {
    float result;
    hipdnnTensorDescriptor_t aDesc, bDesc, resultDesc;
    float alpha = 1.0f, beta = 0.0f;

    // Create tensor descriptors
    hipdnnCreateTensorDescriptor(&aDesc);
    hipdnnCreateTensorDescriptor(&bDesc);
    hipdnnCreateTensorDescriptor(&resultDesc);

    // Set tensor descriptor for a single value
    hipdnnSetTensor4dDescriptor(aDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1);
    hipdnnSetTensor4dDescriptor(bDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1);
    hipdnnSetTensor4dDescriptor(resultDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1);

    // Perform the addition operation
    hipdnnAddTensor(cudnn, &alpha, aDesc, &a, &beta, resultDesc, &result);

    // Destroy descriptors
    hipdnnDestroyTensorDescriptor(aDesc);
    hipdnnDestroyTensorDescriptor(bDesc);
    hipdnnDestroyTensorDescriptor(resultDesc);

    return result;
}

float cudnn_multiply(float a, float b) {
    float result;
    hipdnnTensorDescriptor_t aDesc, bDesc, resultDesc;
    hipdnnOpTensorDescriptor_t opDesc;
    float alpha1 = 1.0f, alpha2 = 1.0f, beta = 0.0f;

    // Create tensor descriptors
    hipdnnCreateTensorDescriptor(&aDesc);
    hipdnnCreateTensorDescriptor(&bDesc);
    hipdnnCreateTensorDescriptor(&resultDesc);

    // Set tensor descriptor for a single value
    hipdnnSetTensor4dDescriptor(aDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1);
    hipdnnSetTensor4dDescriptor(bDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1);
    hipdnnSetTensor4dDescriptor(resultDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1);

    // Create and set operation descriptor
    hipdnnCreateOpTensorDescriptor(&opDesc);
    hipdnnSetOpTensorDescriptor(opDesc, HIPDNN_OP_TENSOR_MUL, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);

    // Perform the multiplication operation
    hipdnnOpTensor(cudnn, opDesc, &alpha1, aDesc, &a, &alpha2, bDesc, &b, &beta, resultDesc, &result);

    // Destroy descriptors
    hipdnnDestroyOpTensorDescriptor(opDesc);
    hipdnnDestroyTensorDescriptor(aDesc);
    hipdnnDestroyTensorDescriptor(bDesc);
    hipdnnDestroyTensorDescriptor(resultDesc);

    return result;
}

float cudnn_relu(float x) {
    float result;
    hipdnnActivationDescriptor_t activationDesc;
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    float alpha = 1.0f, beta = 0.0f;

    // Create tensor descriptors
    hipdnnCreateTensorDescriptor(&inputDesc);
    hipdnnCreateTensorDescriptor(&outputDesc);

    // Set tensor descriptor for a single value
    hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1);
    hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1);

    // Create and set activation descriptor
    hipdnnCreateActivationDescriptor(&activationDesc);
    hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0);

    // Perform the ReLU operation
    hipdnnActivationForward(cudnn, activationDesc, &alpha, inputDesc, &x, &beta, outputDesc, &result);

    // Destroy descriptors
    hipdnnDestroyActivationDescriptor(activationDesc);
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);

    return result;
}

PYBIND11_MODULE(micrograd_cudnn, m) {
    m.def("cudnn_add", &cudnn_add, "A function that adds two numbers using cuDNN");
    m.def("cudnn_multiply", &cudnn_multiply, "A function that multiplies two numbers using cuDNN");
    m.def("cudnn_relu", &cudnn_relu, "A function that applies ReLU using cuDNN");
} 